#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define imin(a,b) (a<b?a:b)

const int N = 1;
const int threadsPerBlock = 1;
const int blocksPerGrid = 1;

__global__ void dot(float *a, float *b, float *c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x; 
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    int i = a[gid];
    int j = b[gid];
    int k = i + j;
    printf("%d", k);
//    if (gid % 2 == 1)
        c[gid] = k;
//    else
//        c[gid] =  1;
//    c[gid] = c[gid] + 1;
}
//__global__ void dot2(float *a, float *b, float*c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x;
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
//}
//__global__ void mykernel(int *data){
//    atomicAdd(data, 10); 
//}
   
int main(){
    float *a, *b, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    
    a = new float[N];
    b = new float[N];
    partial_c = new float[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2.0f;
    }
    hipMalloc((void **)&dev_a, sizeof(float) * N);
    hipMalloc((void **)&dev_b, sizeof(float) * N);
    hipMalloc((void **)&dev_partial_c, N*sizeof(float));
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
    hipMemcpy( partial_c, dev_partial_c, N*sizeof(float), hipMemcpyDeviceToHost);
    #define sum_sq(x) (x*(x+1)*(2*x+1)/6)

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    delete[]  a;
    delete[]  b;
    delete[]  partial_c;
}
